#include "hip/hip_runtime.h"
nclude<stdio.h>
#include<stdlib.h>
#include<limits.h>
#include<hip/hip_runtime.h>
#include"graph.hpp"


template <typename T>
__global__ void initKernel(unsigned V, T* init_array, T initVal)
{
  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id < V)
  {
    init_array[id]=initVal;
  }
}
__global__ void Compute_SSSP_kernel(int * gpu_OA , int * gpu_edgeList , int* gpu_weight, int * gpu_dist , int src ,int V, int MAX_VAL , bool * gpu_modified_prev,
bool * gpu_modified_next, bool * gpu_finished)
{
  unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
  unsigned int v = id;
  if (id < V)
  {
    if (gpu_modified_prev[id] ){
      for (int edge = gpu_OA[id]; edge < gpu_OA[id+1]; edge ++)
      {
        int nbr = gpu_edgeList[edge] ;
        int e = edge;

        int dist_new ;
        if(gpu_dist[id] != MAX_VAL)
          dist_new = gpu_dist[v] + gpu_weight[e];
        //~ int dist_new = gpu_dist[v] + gpu_weight[e];
        if (gpu_dist[nbr] > dist_new)
        //~ if (gpu_dist[id] != MAX_VAL && gpu_dist[nbr] > dist_new)
        {
          atomicMin(&gpu_dist[nbr] , dist_new);
          gpu_modified_next[nbr]=true;
          gpu_finished[0] = false;
        }
      }
    }
  }

}
  void SSSP(int * OA , int * edgeList , int* cpu_edgeLen  , int src ,int V, int E )
{
  int MAX_VAL = 2147483647 ;
  int * gpu_edgeList;
  int * gpu_edgeLen;
  int * gpu_dist;
  int * gpu_OA;
  bool * gpu_modified_prev;
  bool * gpu_modified_next;
  bool * gpu_finished;


  hipMalloc(&gpu_OA,sizeof(int) *(1+V));
  hipMalloc(&gpu_edgeList,sizeof(int) *(E));
  hipMalloc(&gpu_edgeLen,sizeof(int) *(E));
  hipMalloc(&gpu_dist,sizeof(int) *(V));
  hipMalloc(&gpu_modified_prev,sizeof(bool) *(V));
  hipMalloc(&gpu_modified_next,sizeof(bool) *(V));
  hipMalloc(&gpu_finished,sizeof(bool) *(1));

  unsigned int block_size;
  unsigned int num_blocks;
   if(V <= 1024)
   {
    block_size  = V;
    block_size  = 1;
  }
  else
  {
    block_size = 1024;
    num_blocks = ceil(((float)V) / block_size);
  }
  bool * finished = new bool[1];
  finished[0] = false;
  int* dist=new int[V];
  bool* modified=new bool[V];
  for (int t = 0; t < V; t ++)
  {
    dist[t] = INT_MAX;
    modified[t] = false;
  }
  modified[src] = true;
  dist[src] = 0;

  hipEvent_t start, stop; ///TIMER START
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  hipMemcpy (gpu_OA, OA, sizeof(int) *(1+V) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_edgeList, edgeList, sizeof(int) *(E) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_edgeLen, cpu_edgeLen , sizeof(int) *(E) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_dist, dist, sizeof(int) *(V) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_modified_prev, modified , sizeof(bool) *(V) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_finished, finished , sizeof(bool) *(1) ,hipMemcpyHostToDevice);

  int k =0;

  while ( !finished[0] )
  {
    finished[0]=true;//ADDED
    initKernel<bool> <<< 1, 1>>>(1, gpu_finished, true);

    Compute_SSSP_kernel<<<num_blocks , block_size>>>(gpu_OA,gpu_edgeList, gpu_edgeLen ,gpu_dist,src, V ,MAX_VAL , gpu_modified_prev, gpu_modified_next, gpu_finished);
    //~ hipDeviceSynchronize();
    initKernel<bool><<<num_blocks,block_size>>>(V, gpu_modified_prev, false);
    //~ hipDeviceSynchronize();

    hipMemcpy(finished, gpu_finished,  sizeof(bool) *(1), hipMemcpyDeviceToHost); //added this.

    bool *tempModPtr  = gpu_modified_next;
    gpu_modified_next = gpu_modified_prev;
    gpu_modified_prev = tempModPtr;
    ++k;
    if(k==V)
    {
      break;
    }
  }



  hipMemcpy(dist,gpu_dist , sizeof(int) * (V), hipMemcpyDeviceToHost);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms \nIterations:%d\n", milliseconds,k);

   for (int i = 0; i <V; i++)
   {
      printf("%d %d\n", i, dist[i]);
   }


  //~ char *outputfilename = "output_generated.txt";
  //~ FILE *outputfilepointer;
  //~ outputfilepointer = fopen(outputfilename, "w");
  //~ for (int i = 0; i <V; i++)
  //~ {
    //~ fprintf(outputfilepointer, "%d  %d\n", i, dist[i]);
  //~ }

}


// driver program to test above function
int main(int argc , char ** argv)
{
  graph G(argv[1]);
  G.parseGraph();

  int V = G.num_nodes();
//---------------------------------------//
  printf("#nodes:%d\n",V);
//-------------------------------------//
 int E = G.num_edges();

 //---------------------------------------//
  printf("#edges:%d\n",E);
//-------------------------------------//

  int* edgeLen = G.getEdgeLen();

  //~ int* dist;

  int src=0;

  int *OA;
  int *edgeList;
  int *cpu_edgeLen;


   OA = (int *)malloc( (V+1)*sizeof(int));
   edgeList = (int *)malloc( (E)*sizeof(int));
   cpu_edgeLen = (int *)malloc( (E)*sizeof(int));
   //~ dist = (int *)malloc( (V)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = G.indexofNodes[i];
    OA[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = G.edgeList[i];
    edgeList[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = edgeLen[i];
    cpu_edgeLen[i] = temp;
  }


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    SSSP(OA,edgeList, cpu_edgeLen ,src, V,E);
    //~ hipDeviceSynchronize();

    //~ hipEventRecord(stop,0);
    //~ hipEventSynchronize(stop);
    //~ hipEventElapsedTime(&milliseconds, start, stop);
    //~ printf("Time taken by function to execute is: %.6f ms\n", milliseconds);


  return 0;

}

