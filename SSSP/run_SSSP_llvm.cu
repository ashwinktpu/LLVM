#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        const char *errStr; \
        hipDrvGetErrorString(err, &errStr); \
        fprintf(stderr, "CUDA Error: %s\n", errStr); \
        exit(EXIT_FAILURE); \
    } \
}

int main() {
    // Initialize the CUDA driver
    CHECK_CUDA_ERROR(hipInit(0));

    // Get the first CUDA device
    hipDevice_t device;
    CHECK_CUDA_ERROR(hipDeviceGet(&device, 0));

    // Create a CUDA context
    hipCtx_t context;
    CHECK_CUDA_ERROR(hipCtxCreate(&context, 0, device));

    // Load the PTX file
    hipModule_t module;
    CHECK_CUDA_ERROR(hipModuleLoad(&module, "SSSP.ptx"));

    // Get the kernel functions
    hipFunction_t init_func;
    hipFunction_t sssp_func;
    CHECK_CUDA_ERROR(hipModuleGetFunction(&init_func, module, "initKernel"));
    CHECK_CUDA_ERROR(hipModuleGetFunction(&sssp_func, module,"Compute_SSSP_kernel"));

    // Sample graph data (replace with actual graph input if needed)
    int V = 5;
    int E = 8;
    int OA[] = {0, 2, 4, 5, 7, 8};
    int edgeList[] = {1, 2, 0, 3, 2, 1, 4, 3};
    int weight[] = {2, 4, 1, 7, 3, 2, 1, 5};
    int src = 0;

    int MAX_VAL = INT_MAX;
    int *dist = (int*)malloc(V * sizeof(int));
    bool *modified_prev = (bool*)malloc(V * sizeof(bool));
    bool *modified_next = (bool*)malloc(V * sizeof(bool));
    bool finished = false;

    // Initialize arrays
    for (int i = 0; i < V; ++i) {
        dist[i] = MAX_VAL;
        modified_prev[i] = false;
        modified_next[i] = false;
    }
    dist[src] = 0;
    modified_prev[src] = true;

    // Allocate device memory
    hipDeviceptr_t gpu_OA, gpu_edgeList, gpu_weight, gpu_dist, gpu_modified_prev, gpu_modified_next, gpu_finished;
    CHECK_CUDA_ERROR(hipMalloc(&gpu_OA, sizeof(int) * (V + 1)));
    CHECK_CUDA_ERROR(hipMalloc(&gpu_edgeList, sizeof(int) * E));
    CHECK_CUDA_ERROR(hipMalloc(&gpu_weight, sizeof(int) * E));
    CHECK_CUDA_ERROR(hipMalloc(&gpu_dist, sizeof(int) * V));
    CHECK_CUDA_ERROR(hipMalloc(&gpu_modified_prev, sizeof(bool) * V));
    CHECK_CUDA_ERROR(hipMalloc(&gpu_modified_next, sizeof(bool) * V));
    CHECK_CUDA_ERROR(hipMalloc(&gpu_finished, sizeof(bool)));

    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpyHtoD(gpu_OA, OA, sizeof(int) * (V + 1)));
    CHECK_CUDA_ERROR(hipMemcpyHtoD(gpu_edgeList, edgeList, sizeof(int) * E));
    CHECK_CUDA_ERROR(hipMemcpyHtoD(gpu_weight, weight, sizeof(int) * E));
    CHECK_CUDA_ERROR(hipMemcpyHtoD(gpu_dist, dist, sizeof(int) * V));
    CHECK_CUDA_ERROR(hipMemcpyHtoD(gpu_modified_prev, modified_prev, sizeof(bool) * V));

    // Set kernel launch parameters
    int block_size = 1024;
    int num_blocks = (V + block_size - 1) / block_size;

    // Run the SSSP algorithm
    do {
        finished = true;
        CHECK_CUDA_ERROR(hipMemcpyHtoD(gpu_finished, &finished, sizeof(bool)));

       // void *args1[] = {&gpu_finished, &V, &gpu_modified_prev, &modified_prev[0]};
        void *args1[] = { &V, &gpu_modified_prev, &modified_prev[0]};
        void *args2[] = {&gpu_OA, &gpu_edgeList, &gpu_weight, &gpu_dist, &V, &MAX_VAL, &gpu_modified_prev, &gpu_modified_next, &gpu_finished};

        CHECK_CUDA_ERROR(hipModuleLaunchKernel(init_func, 1, 1, 1, 1, 1, 1, 0, 0, args1, 0));
        CHECK_CUDA_ERROR(hipModuleLaunchKernel(sssp_func, num_blocks, 1, 1, block_size, 1, 1, 0, 0, args2, 0));

        CHECK_CUDA_ERROR(hipMemcpyDtoH(&finished, gpu_finished, sizeof(bool)));

        // Swap pointers
        hipDeviceptr_t temp = gpu_modified_prev;
        gpu_modified_prev = gpu_modified_next;
        gpu_modified_next = temp;
    } while (!finished);

    // Copy results back to host
    CHECK_CUDA_ERROR(hipMemcpyDtoH(dist, gpu_dist, sizeof(int) * V));

    // Print results
    for (int i = 0; i < V; ++i) {
        printf("Node %d: Distance = %d\n", i, dist[i]);
    }

    // Clean up
    hipFree(gpu_OA);
    hipFree(gpu_edgeList);
    hipFree(gpu_weight);
    hipFree(gpu_dist);
    hipFree(gpu_modified_prev);
    hipFree(gpu_modified_next);
    hipFree(gpu_finished);
    hipModuleUnload(module);
    hipCtxDestroy(context);

    free(dist);
    free(modified_prev);
    free(modified_next);

    return 0;
}

