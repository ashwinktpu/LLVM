#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA_ERROR(call) {                                \
    hipError_t err = call;                                        \
    if (err != hipSuccess) {                                 \
        const char *errStr;                                    \
        hipDrvGetErrorString(err, &errStr);                       \
        fprintf(stderr, "CUDA Error: %s\n", errStr);          \
        exit(EXIT_FAILURE);                                   \
    }                                                          \
}

int main() {
    // Initialize the CUDA driver API
    CHECK_CUDA_ERROR(hipInit(0));

    // Get the first CUDA device
    hipDevice_t device;
    CHECK_CUDA_ERROR(hipDeviceGet(&device, 0));

    // Create a CUDA context
    hipCtx_t context;
    CHECK_CUDA_ERROR(hipCtxCreate(&context, 0, device));

    // Load the PTX file
    hipModule_t module;
    CHECK_CUDA_ERROR(hipModuleLoad(&module, "demo.ptx"));

    // Get the kernel function
    hipFunction_t add_func;
    CHECK_CUDA_ERROR(hipModuleGetFunction(&add_func, module, "add"));

    // Allocate device memory
    int c = 0;
    hipDeviceptr_t d_c;
    CHECK_CUDA_ERROR(hipMalloc(&d_c, sizeof(int)));

    // Set kernel parameters
    int a = 1, b = 2;
    void *args[] = { &d_c, &a, &b };

    // Launch the kernel
    CHECK_CUDA_ERROR(hipModuleLaunchKernel(
        add_func,
        1, 1, 1,     // Grid dimensions
        1, 1, 1,     // Block dimensions
        0, 0,        // Shared memory size and stream
        args, 0      // Kernel arguments and extra options
    ));

    // Copy the result back to the host
    CHECK_CUDA_ERROR(hipMemcpyDtoH(&c, d_c, sizeof(int)));

    // Print the result
    printf("Result: %d\n", c);

    // Clean up
    hipFree(d_c);
    hipModuleUnload(module);
    hipCtxDestroy(context);

    return 0;
}

